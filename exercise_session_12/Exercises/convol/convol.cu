#include <time.h>
#include <hip/hip_runtime.h>
#include <stdio.h>
#define STOP 0
#define START 1

#define BLOCKSIZE 256


extern "C" void chrono (int kind, float *time);

__global__ void kconvol (float *gpu_a, float *gpu_b, int n) {
  int i, j, l;
  // TO DO : evaluate the global 1D index l of the current thread,
  // using blockDim, blockIdx and threadIdx.
  l = threadIdx.x + blockIdx.x * blockDim.x;
  int bIdx = (blockIdx.x);
  int cDim = (blockDim.x);
  int TIdx = (threadIdx.x);
  
  // TO DO : evaluate global indices of thread (i,j) from the index l
  j = l % n;
  i = l / n;
  //printf("%d\n", l);
  //printf("---------\nl = %d, (i,j) = (%d,%d)\nblockIdx: %d\nblockDim: %d\nthreadIdx: %d\n", l, i, j, bIdx, cDim, TIdx);

  if ((i >= n) || (j >= n)) return;
  if ((i == 0) || (j == 0) || (i == n-1) || (j == n-1))  {
    gpu_b[l] = gpu_a[l]; // edges are untouched
  }
  else
    // TO DO : fill up the MISSING indices below
    gpu_b[l]=(1./5.)*(gpu_a[l-n] + gpu_a[l-1] + gpu_a[l] + gpu_a[l+1]+ gpu_a[l+n]);
}

extern "C" void gpu_convol (float *a, float *b, int n, int blocks) {
  float *gpu_a;
  float *gpu_b;
  hipError_t err;
  float time;
  
  err = hipMalloc (&gpu_a, n*n*sizeof(float));
  if (err != 0) {
    printf ("Error allocating gpu_a: %s\n", hipGetErrorString (err));
    exit (1);
  }
  err = hipMalloc (&gpu_b, n*n*sizeof(float));
  if (err != 0) {
    printf ("Error allocating gpu_b: %s\n", hipGetErrorString (err));
    exit (1);
  }
  
  hipMemcpy (gpu_a, a, n*n*sizeof(float), hipMemcpyHostToDevice);
    
  // NOTE : the chronometer below does not contemplate overhead of memory allocation and
  // memory transfer.
  chrono (START, &time);
  // TO DO : the number of blocks is missing below in the kernel invocation
  //int blocks = (1000192) / BLOCKSIZE;
  printf("block: %d\n", blocks);
  printf("blocksize: %d\n", BLOCKSIZE);
  kconvol <<<blocks,BLOCKSIZE>>> (gpu_a, gpu_b, n);
  err=hipDeviceSynchronize ();
  chrono (STOP, &time);
  printf ("Convolution took  %f sec. on GPU\n", time);
  hipMemcpy (b, gpu_b, n*n*sizeof(float), hipMemcpyDeviceToHost);
  if (err != 0) {
    printf ("%s\n", hipGetErrorString (err));
    exit (1);
  }
  FILE *fp;
  fp = fopen("timing_plot_1000x1000.out", "a");
  fprintf(fp, "%d, %.10g\n", blocks, time);
  hipFree (gpu_a);
  hipFree (gpu_b);
}
