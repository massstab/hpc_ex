#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <math.h>

#define BLOCKSIZE 256

__global__ void kern_set_val (float *gpu_ptr, float value, int n) {
  int i;
  //TO DO: evaluate the value of i
  i = threadIdx.x + blockIdx.x * blockDim.x;
  gpu_ptr[i] = value;
}

int main () {
  int i, failed=0;
  int N = 1024;   // size of vector
  float *ptr;     // Host pointer 
  float *gpu_ptr; // Device pointer
  
  /* Allocate vector in Host*/
  ptr = (float *)malloc(sizeof(float)*N);
  /* Allocate vector in Device*/
  hipMalloc (&gpu_ptr, sizeof(float)*N);

  //TO DO : write kernel invocation here
  kern_set_val<<<(N)/BLOCKSIZE,BLOCKSIZE>>>(gpu_ptr, 11, N);

  hipDeviceSynchronize ();

  //TO DO : copy data to host
  hipMemcpy(ptr, gpu_ptr, N*sizeof(float), hipMemcpyDeviceToHost);

  hipFree (gpu_ptr);

  /* Now check that it did what we want */

  for (i = 0; i < 10; i++)//first ten values are written
    printf ("%f\t", ptr[i]);
  printf ("\n");
  for (i = N-10; i < N; i++)//last ten values are written
    printf ("%f\t", ptr[i]);
  printf ("\n");

  for (i = 0; i < N; i++) {//All values are compared
    if (fabs(ptr[i]-11.0) > 1e-8) {
      failed=1;
    }
  }
  if (failed) {
    printf ("FAILED !!\n");
  } else {
    printf ("PASSED !!\n");
  }    
  free (ptr);
}
