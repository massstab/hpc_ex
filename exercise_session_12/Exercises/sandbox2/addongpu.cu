#include <hip/hip_runtime.h>
#include <stdlib.h>
#include <stdio.h>
#include <math.h>

/* Define the matrix size */
#define NX 200
#define NY 100

/* Size of a block */
#define BLOCKSIZE 256

__global__ void kernadd (float* mout, float* min1, float *min2, int nx, int ny)
{
  int i, j, index;
  index = blockDim.x*blockIdx.x+threadIdx.x;
  j = index/nx;
  i = index - j*nx;
  if ((i < nx) && (j < ny))
    mout[index] = min1[index] + min2[index];
    
}


/*******************************************************/
/*  We initialize the vectors with random values       */
/*******************************************************/

void Init(float* mat, int nx, int ny) {
  int i, j;
  for (i = 0; i < nx; i++) {	/* 2D loop */
    for (j = 0; j < ny; j++) {
      mat[i+j*nx] = drand48 ();	/* position of cell (i,j) */
    }
  }
}


/*******************************************************/
/*            MAIN PROGRAM                             */
/*******************************************************/

int main () {
  int i=0, error=0, nx=NX, ny=NY;
  float diff;
  int size = nx * ny * sizeof(float);

  /* Matrix allocation */
  float *mat_in1 = (float*) malloc(size);
  float *mat_in2 = (float*) malloc(size);
  float *mat_out = (float*) malloc(size);

  /* Matrix allocation on device */
  float *mat_out_gpu, *mat_in1_gpu, *mat_in2_gpu;
  /* TO DO : do the allocation below, using cudaMalloc()*/
  hipMalloc (&mat_in1_gpu, size);
  hipMalloc (&mat_in2_gpu, size);
  hipMalloc (&mat_out_gpu, size);

  /* Matrix initialization */
  Init(mat_in1, nx, ny);
  Init(mat_in2, nx, ny);  
  
  /* TO DO : write below the instructions to copy it to the device */
  hipMemcpy(mat_in1_gpu, mat_in1, size, hipMemcpyHostToDevice);
  hipMemcpy(mat_in2_gpu, mat_in2, size, hipMemcpyHostToDevice);
  hipMemcpy(mat_out_gpu, mat_out, size, hipMemcpyHostToDevice);
  
  /* TO DO : complete the number of blocks below */
  int numBlocks = (nx * ny + BLOCKSIZE-1) / BLOCKSIZE;
 
  /* TO DO : kernel invocation */
  kernadd<<<numBlocks, BLOCKSIZE>>>(mat_out_gpu, mat_in1_gpu, mat_in2_gpu, nx, ny);

  hipDeviceSynchronize();
  
  /* We now transfer back the matrix from the device to the host */
  /* TO DO : write cudaMemcpy() instruction below */
  hipMemcpy(mat_out, mat_out_gpu, size, hipMemcpyDeviceToHost);  
    
  /* free memory */
  hipFree(mat_out_gpu);
  hipFree(mat_in1_gpu);
  hipFree(mat_in2_gpu);

  /* We now check that the result is correct */

  for (i=0; i< nx*ny; i++) {	/* No need for a 2D loop, actually ! */
    diff = mat_out[i] - (mat_in1[i]+mat_in2[i]);
    if (fabs(diff) > 0.0000001f) {
      error = 1;
    }
  }

  if (error) {
    printf("FAILED\n");
  }
  else {
    printf("PASSED\n");
  }
  free (mat_in1);
  free (mat_in2);
  free (mat_out);
}



